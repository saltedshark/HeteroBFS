#include "hip/hip_runtime.h"
// src/bfs_cuda.cu
// ================================
// 该代码使用了[Altis](https://github.com/utcs-scea/altis)代码并对其进行了部分修改
// Copyright (c) 2021, Bodun Hu, et al.
// [BSD-2条款]许可证见本项目根目录的licenses/altis.txt
// ================================

/***
 * 函数功能：使用cuda并行的bfs对图进行遍历，不管图有没有连通
 * 将普通cuda版本和uvm，uvm-advise，uvm-prefetch，uvm-advise-prefetch集成到一起
 * 本程序使用传统的层级同步方法，逐层遍历图，每一层处理完后同步所有线程，再进入下一层。
 * 特点：
 *  适合 CPU：层间同步容易实现（如 OpenMP barrier）
 *  缺点：GPU 上因多次内核启动和全局同步（hipDeviceSynchronize）导致高开销
 */
#include <stdio.h>
#include <cstdlib>
#include <string>
using std::string;
#include <chrono>
using namespace std::chrono;
#include <iostream>
#include <queue>
#include <fstream>
#include <cstdint>
#include <iomanip>// 用于十六进制输出
#include <cstring> //for memset

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "OptionParser.h"//for arguments parse
#include "cudacommon.h" //for checkCudaErrors，assert
#include <cfloat>//for FLT_MAX

#include <thrust/find.h>
#include <thrust/execution_policy.h>

// CSR二进制文件头（兼容Gunrock）
struct CSRHeader {
    uint32_t magic;      // 魔数校验
    uint32_t num_nodes;  // 节点数
    uint32_t num_edges;  // 边数
    uint32_t _padding=0;   // 填充字段
};

// 打印十六进制值的辅助函数
void PrintHex(const char* label, uint32_t value) {
    std::cout << std::left << std::setw(15) << label 
              << "0x" << std::hex << std::uppercase 
              << std::setw(8) << std::setfill('0') << value << std::endl;
}

void checkCudaFeatureAvailability(OptionParser &op);

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	BFS graph runner. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/19/2020. </remarks>
///
/// <param name="resultDB">		 	[in,out] The result database. </param>
/// <param name="op">			 	[in,out] The operation. </param>
/// <param name="no_of_nodes">   	The no of nodes. </param>
/// <param name="edge_list_size">	Size of the edge list. </param>
/// <param name="source">		 	Source for the. </param>
/// <param name="h_graph_nodes"> 	[in,out] [in,out] If non-null, the graph nodes. </param>
/// <param name="h_graph_edges"> 	[in,out] [in,out] If non-null, the graph edges. </param>
///
/// <returns>	A float. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

float BFSGraph(hipDeviceProp_t &deviceProp, int no_of_nodes, int edge_list_size, uint32_t *&offsets, uint32_t *&edges);

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	BFS graph using unified memory. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/19/2020. </remarks>
///
/// <param name="resultDB">		 	[in,out] The result database. </param>
/// <param name="op">			 	[in,out] The operation. </param>
/// <param name="no_of_nodes">   	The no of nodes. </param>
/// <param name="edge_list_size">	Size of the edge list. </param>
/// <param name="source">		 	Source for the. </param>
/// <param name="h_graph_nodes"> 	[in,out] [in,out] If non-null, the graph nodes. </param>
/// <param name="h_graph_edges"> 	[in,out] [in,out] If non-null, the graph edges. </param>
///
/// <returns>	A float. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

float BFSGraphUnifiedMemory(OptionParser &op, hipDeviceProp_t &deviceProp, int no_of_nodes, int edge_list_size, uint32_t *&offsets, uint32_t *&edges);

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	BFS Kernel. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/19/2020. </remarks>
///
/// <param name="g_graph_nodes">			[in,out] If non-null, the graph nodes. </param>
/// <param name="g_graph_edges">			[in,out] If non-null, the graph edges. </param>
/// <param name="g_graph_mask">				[in,out] If non-null, true to graph mask. </param>
/// <param name="g_updating_graph_mask">	[in,out] If non-null, true to updating graph mask. </param>
/// <param name="g_graph_visited">			[in,out] If non-null, true if graph visited. </param>
/// <param name="g_cost">					[in,out] If non-null, the cost. </param>
/// <param name="no_of_nodes">				The no of nodes. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Kernel( uint32_t *g_offsets, uint32_t *g_edges, bool* g_graph_mask, 
    bool* g_updating_graph_mask, bool *g_graph_visited, int* g_cost, int no_of_nodes) 
{
    //int tid = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.y + threadIdx.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid<no_of_nodes && g_graph_mask[tid])
	{
		g_graph_mask[tid]=false;
        uint32_t start = g_offsets[tid];
        uint32_t end = g_offsets[tid + 1];
		for(int i = start; i < end; i++){
            uint32_t neighbor = g_edges[i];
			if(!g_graph_visited[neighbor]){
				g_cost[neighbor]=g_cost[tid]+1;
				g_updating_graph_mask[neighbor]=true;
			}
		}
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	BFS Kernel 2. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/19/2020. </remarks>
///
/// <param name="g_graph_mask">				[in,out] If non-null, true to graph mask. </param>
/// <param name="g_updating_graph_mask">	[in,out] If non-null, true to updating graph mask. </param>
/// <param name="g_graph_visited">			[in,out] If non-null, true if graph visited. </param>
/// <param name="g_over">					[in,out] If non-null, true to over. </param>
/// <param name="no_of_nodes">				The no of nodes. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Kernel2( bool* g_graph_mask, bool *g_updating_graph_mask,
     bool* g_graph_visited, bool *g_over, int no_of_nodes)
{
    //int tid = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.y + threadIdx.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid<no_of_nodes && g_updating_graph_mask[tid])
	{
		g_graph_mask[tid]=true;
		g_graph_visited[tid]=true;
		*g_over=true;
		g_updating_graph_mask[tid]=false;
	}
}

//initGraph负责从csr.bin的文件内读取信息，校验后从文件头获取节点数、边数信息(这里的边数已经是edges数组大小，无需再乘以2)，然后读取offset和edges数组
void initGraph(const string &filename, int &no_of_nodes, int &edge_list_size, uint32_t *&offsets, uint32_t *&edges);

void cuda_bfs(int no_of_nodes, int source,
    uint32_t *&d_offsets, uint32_t *&d_edges, bool* &d_graph_mask, bool* &d_updating_graph_mask, 
    bool* &d_graph_visited, int* &d_cost, bool *&d_over, 
    dim3 &grid, dim3 &block, double &kernel_time, double &transfer_time, int &k);

void cuda_bfs_uvm(int no_of_nodes, int source, uint32_t *&graph_offsets, uint32_t *&graph_edges,
        bool* &graph_mask, bool* &updating_graph_mask, bool* &graph_visited, int* &cost, bool *&over, 
        dim3 &grid, dim3 &block, double &kernel_time, int &k);

void opinit(OptionParser &op);

//main内根据参数解析判断是否使用uvm相关，调用不同的BFSGraph,BFSGraphUnifiedMemory
int main(int argc, char** argv){
    //参数预设置
    // Get args
    OptionParser op;
    opinit(op);
    
    if (!op.parse(argc, argv))
    {
        op.usage();
        return (op.HelpRequested() ? 0 : 1);
    }

    // Check CUDA feature availability
    checkCudaFeatureAvailability(op);

    //设备设置
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    //解析本程序所用参数
    string filename = op.getOptionString("inputFile");
    int passes = op.getOptionInt("passes");
    bool quiet = op.getOptionBool("quiet");
    const bool uvm = op.getOptionBool("uvm");
    const bool uvm_advise = op.getOptionBool("uvm-advise");
    const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
    const bool uvm_prefetch_advise = op.getOptionBool("uvm-prefetch-advise");

	int no_of_nodes = 0;
	int edge_list_size = 0;
    uint32_t *offsets;
    uint32_t *edges;

    //读文件获取信息
	initGraph(filename, no_of_nodes, edge_list_size, offsets, edges);

    //执行passes次，根据参数判断使用啥程序，普通还是uvm相关
    printf("Running bfs_cuda\n");
	for (int i = 0; i < passes; i++) {
        if (!quiet) {
            printf("Pass %d:\n", i);
        }
        if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
            //调用uvm相关函数
            float timeUM = BFSGraphUnifiedMemory(op, deviceProp, no_of_nodes, edge_list_size, offsets, edges);
            if (!quiet) {
                if (timeUM == FLT_MAX) {
                    printf("Executing BFS using unified memory...Error.\n");
                } else {
                    printf("Executing BFS using unified memory...Done.\n");
                }
            }
        } else {
            //调用普通cuda执行
            float time = BFSGraph(deviceProp, no_of_nodes, edge_list_size,  offsets, edges);
            if (!quiet) {
                if (time == FLT_MAX) {
                    printf("Executing BFS...Error.\n");
                } else {
                    printf("Executing BFS...Done.\n");
                }
            }
        }
    }

    //清理内存,这个内存分配发生在initGraph内
    free(offsets);
    free(edges);
	return 0;
}

void opinit(OptionParser &op){
    // Add shared options to the parser
    op.addOption("passes", OPT_INT, "10", "specify number of passes", 'n');
    op.addOption("verbose", OPT_BOOL, "0", "enable verbose output", 'v');
    op.addOption("quiet", OPT_BOOL, "0", "enable concise output", 'q');
    op.addOption("inputFile", OPT_STRING, "", "path of input file", 'i');
    // op.addOption("outputFile", OPT_STRING, "", "path of output file", 'o');
    op.addOption("device", OPT_VECINT, "0", "specify device(s) to run on", 'd');

    // Add options for turn on/off CUDA features
    op.addOption("uvm", OPT_BOOL, "0", "enable CUDA Unified Virtual Memory, only demand paging");
    op.addOption("uvm-advise", OPT_BOOL, "0", "guide the driver about memory usage patterns");
    op.addOption("uvm-prefetch", OPT_BOOL, "0", "prefetch memory the specified destination device");
    op.addOption("uvm-prefetch-advise", OPT_BOOL, "0", "prefetch memory the specified destination device with memory guidance on");
    op.addOption("coop", OPT_BOOL, "0", "enable CUDA Cooperative Groups");
    op.addOption("dyn", OPT_BOOL, "0", "enable CUDA Dynamic Parallelism");
    op.addOption("graph", OPT_BOOL, "0", "enable CUDA Graphs");
}

void initGraph(const string &filename, int &no_of_nodes, int &edge_list_size, uint32_t *&offsets, uint32_t *&edges){
	
    std::ifstream ifs(filename, std::ios::binary);
    if (!ifs) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        exit(0);
    }

	printf("Reading graph file\n");
    // 读取文件头
    CSRHeader header;
    ifs.read(reinterpret_cast<char*>(&header), sizeof(CSRHeader));
    // 校验魔数
    if (header.magic != 0x47535246) {  // "GSRF" 的十六进制
        std::cerr << "文件格式错误，无效的魔数" << std::endl;
        PrintHex("预期魔数", 0x47535246);
        PrintHex("实际魔数", header.magic);
        exit(0);
    }
    // 校验数据合理性
    if (header.num_nodes == 0 || header.num_edges == 0) {
        std::cerr << "文件头数据异常: 节点数=" << header.num_nodes 
                  << " 边数=" << header.num_edges << std::endl;
        exit(0);
    }
    // 输出头信息
    std::cout << "====== 文件头信息 ======" << std::endl;
    PrintHex("魔数", header.magic);
    std::cout << std::dec;  // 切回十进制输出
    std::cout << "节点数: " << header.num_nodes << std::endl;
    std::cout << "边数: " << header.num_edges << std::endl;
    std::cout << "填充字段: " << header._padding << std::endl;

    no_of_nodes = header.num_nodes;
    edge_list_size = header.num_edges;

    // offsets.reserve(header.num_nodes + 1);
    // edges.reserve(header.num_edges);
    offsets = (uint32_t*) malloc(sizeof(uint32_t) * (no_of_nodes+1));//offset需要多分配1个
    edges = (uint32_t*) malloc(sizeof(uint32_t) * edge_list_size);

    // 读取偏移数组
    ifs.read(reinterpret_cast<char*>(offsets), 
            (no_of_nodes + 1) * sizeof(uint32_t));

    // 读取边索引
    ifs.read(reinterpret_cast<char*>(edges), 
            edge_list_size * sizeof(uint32_t));

    // 验证读取完整性
    if (!ifs) {
        std::cerr << "文件读取不完整或已损坏" << std::endl;
        exit(0);
    }
	
	std::cout << "\n文件验证通过，数据结构完整" << std::endl;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Bfs graph using CUDA. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/19/2020. </remarks>
///
/// <param name="resultDB">		 	[in,out] The result database. </param>
/// <param name="op">			 	[in,out] The operation. </param>
/// <param name="no_of_nodes">   	The no of nodes. </param>
/// <param name="edge_list_size">	Size of the edge list. </param>
/// <param name="source">		 	Source for the. </param>
/// <param name="h_graph_nodes"> 	[in,out] [in,out] If non-null, the graph nodes. </param>
/// <param name="h_graph_edges"> 	[in,out] [in,out] If non-null, the graph edges. </param>
///
/// <returns>	Transfer time and kernel time. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

// 查找未访问节点的辅助函数
int find_unvisited_vertex(bool* d_visited, int num_vertices) {
    auto start = d_visited;
    auto end = d_visited + num_vertices;
    auto iter = thrust::find_if(
        thrust::device,
        start,
        end,
        [] __device__ (bool v) { return !v; }
    );
    //首先通过find_if查找未访问节点
    //然后判断位置是否有效再返回值
    return (iter != end) ? (iter - start): -1 ;
}

float BFSGraph(hipDeviceProp_t &deviceProp, int no_of_nodes, int edge_list_size, uint32_t *&offsets, uint32_t *&edges) 
{
    //该函数功能：
    //分配状态相关以及存结果内存并初始化
    //分配设备内存，避免每次都在内层函数初始化，外层分配一次，内层只涉及多次拷贝
        //部分数据拷贝一次全局即可，如h_graph_nodes、h_graph_edges以及h_updating_graph_mask，每次调用内层循环都不会更改
        //部分数据需要全局统一拷贝一次，之后用的时候再拷贝具体数据即可，避免大批量数据拷贝，如h_graph_mask,h_graph_visited,h_cost
    //设置执行环境+参数，避免内层循环多次设置
    //调用内层循环执行
    //结果拷贝回来，只需要拷贝d_cost回来即可，其余用不到
    //释放内存,设备+主机
    //时间统一记录
    //返回函数值
    auto start_t = high_resolution_clock::now();

    double total_time = 0;
    double kernel_time = 0;
    double transfer_time = 0;
    
    int *h_cost = (int*) malloc( sizeof(int)*no_of_nodes);
    assert(h_cost);
    memset(h_cost, -1, no_of_nodes * sizeof(int));

    //设备内存
    uint32_t *d_offsets = nullptr;
    uint32_t *d_edges = nullptr;
	// mask
	bool* d_graph_mask = nullptr;
	bool* d_updating_graph_mask = nullptr;
	// visited nodes
	bool* d_graph_visited = nullptr;
    // result
	int* d_cost = nullptr;
	// bool if execution is over
	bool *d_over = nullptr;

    //统一分配
    //CUDA_SAFE_CALL_NOEXIT宏定义于cudacommon
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_offsets, sizeof(uint32_t) * (no_of_nodes+1)));//注意offset要多分配一个
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_edges, sizeof(uint32_t) * edge_list_size));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_graph_mask, sizeof(bool)*no_of_nodes));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_updating_graph_mask, sizeof(bool)*no_of_nodes));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_graph_visited, sizeof(bool)*no_of_nodes));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_cost, sizeof(int)*no_of_nodes));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_over, sizeof(bool)));
    hipError_t err = hipGetLastError();

    if(err != hipSuccess) {
        free( h_cost);
        hipFree(d_offsets);
        hipFree(d_edges);
        hipFree(d_graph_mask);
        hipFree(d_updating_graph_mask);
        hipFree(d_graph_visited);
        hipFree(d_cost);  
        hipFree(d_over);
        return FLT_MAX;
    }

    hipEvent_t tstart, tstop;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);
    float elapsedTime = 0;
    hipEventRecord(tstart, 0);
    //nodes和edges拷贝一次就行，这个不会变的，因此放外层函数拷贝
    hipMemcpy(d_offsets, offsets, sizeof(uint32_t)*(no_of_nodes+1), hipMemcpyHostToDevice);
    hipMemcpy(d_edges, edges, sizeof(uint32_t)*edge_list_size, hipMemcpyHostToDevice);
    hipEventRecord(tstop, 0);
    hipEventSynchronize(tstop);
    hipEventElapsedTime(&elapsedTime, tstart, tstop);
    transfer_time += elapsedTime * 1.e-3; // convert to seconds

    //直接在设备端初始化，省去创建主机数组、拷贝
    hipMemset(d_updating_graph_mask, 0, no_of_nodes * sizeof(bool));
    hipMemset(d_graph_mask, 0, no_of_nodes * sizeof(bool));
    hipMemset(d_graph_visited, 0, no_of_nodes * sizeof(bool));
    hipMemset(d_cost, 0xFF, no_of_nodes * sizeof(int));//初始化为-1
    

    //内层循环执行参数设置
    int num_of_blocks = 1;
    int num_of_threads_per_block = no_of_nodes;//正常限制为1024，这里先设置为节点数，然后再计算
    int max_threads_per_block = deviceProp.maxThreadsPerBlock;//硬件限制的
    //Make execution Parameters according to the number of nodes
    //Distribute threads across multiple Blocks if necessary
    if (no_of_nodes > max_threads_per_block)
    {
        num_of_blocks = (int)ceil(no_of_nodes/(double)max_threads_per_block);
        num_of_threads_per_block = max_threads_per_block; 
    }
    //setup execution parameters
    dim3  grid( num_of_blocks, 1, 1);//1维grid，有num_of_blocks个block
    dim3  block( num_of_threads_per_block, 1, 1);//1维threads

	//遍历所有节点，未访问就进入遍历
    int cnt = 0;//用于记录连通块数量
    int k = 0; //记录kernel执行次数
    
    //使用thrust在设备端查找，不用反复拷贝visited数组
    while(true){
        //设备端查找未访问节点
        int32_t start_vertex = find_unvisited_vertex(d_graph_visited, no_of_nodes);
        // printf("start node is %d\n", start_vertex);
        if(start_vertex == -1) break;//全访问过
        //然后调用cuda_bfs执行
        // printf("visite node : %d\n", i);
        cuda_bfs(no_of_nodes, start_vertex,
            d_offsets, d_edges, d_graph_mask, d_updating_graph_mask, 
            d_graph_visited, d_cost, d_over, grid, block, kernel_time, transfer_time, k);

            ++cnt;//连通块统计

    }

    //统一拷贝回值
    //只有d_cost拷贝回传
    hipEventRecord(tstart, 0);
    hipMemcpy(h_cost, d_cost, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost);
    hipEventRecord(tstop, 0);
    hipEventSynchronize(tstop);
    hipEventElapsedTime(&elapsedTime, tstart, tstop);
    transfer_time += elapsedTime * 1.e-3; // convert to seconds

    auto end_t = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(end_t - start_t);
    double duration_t = double(duration.count()) * microseconds::period::num / microseconds::period::den;
    total_time += duration_t;


    //事件销毁
    hipEventDestroy(tstart);
    hipEventDestroy(tstop);
    //clean device mem
    hipFree(d_offsets);
	hipFree(d_edges);
	hipFree(d_graph_mask);
	hipFree(d_updating_graph_mask);
	hipFree(d_graph_visited);
	hipFree(d_cost);  
    hipFree(d_over);

	//cleanup memory
	free( h_cost);
	
    //时间统输出一记录
    printf("Time record(seconds)\n");
    printf("total_time : %f\n", total_time);
    printf("transfer_time : %f\n", transfer_time);
    printf("kernel_time : %f\n", kernel_time);
    printf("graph_block : %d\n", cnt);
    printf("kernel_exe_times : %d\n", k);

    return total_time;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Bfs graph with unified memory using CUDA. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/19/2020. </remarks>
///
/// <param name="resultDB">		 	[in,out] The result database. </param>
/// <param name="op">			 	[in,out] The operation. </param>
/// <param name="no_of_nodes">   	The no of nodes. </param>
/// <param name="edge_list_size">	Size of the edge list. </param>
/// <param name="source">		 	Source for the. </param>
/// <param name="h_graph_nodes"> 	[in,out] [in,out] If non-null, the graph nodes. </param>
/// <param name="h_graph_edges"> 	[in,out] [in,out] If non-null, the graph edges. </param>
///
/// <returns>	Kernel time. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

float BFSGraphUnifiedMemory(OptionParser &op, hipDeviceProp_t &deviceProp, int no_of_nodes, int edge_list_size, uint32_t *&offsets, uint32_t *&edges) {
    //该函数功能:
        //获取输入参数
        //设置执行环境+参数，避免内层循环多次设置
        //统一内存相关操作
            //分配nodes、edges的统一内存，并从主存拷贝到统一内存，然后根据输入参数进行相应操作
            //分配状态相关打3个统一内存数组以及存储结果打cost统一内存数据，初始化后，根据输入参数做相应操作
            //为over分配统一内存
        //调用内层循环进行执行
        //拷贝回结果数组cost到cpu
        //释放内存
        //统一时间记录
        //返回函数值


    auto start_t = high_resolution_clock::now();

    //获取输入参数
    // bool verbose = op.getOptionBool("verbose");
    // bool quiet = op.getOptionBool("quiet");
    int device = op.getOptionInt("device");
    const bool uvm = op.getOptionBool("uvm");
    const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
    const bool uvm_advise = op.getOptionBool("uvm-advise");
    const bool uvm_prefetch_advise = op.getOptionBool("uvm-prefetch-advise");

    double total_time = 0;
    double kernel_time = 0;
    // double transfer_time = 0;

    //内层循环执行参数设置
    int num_of_blocks = 1;
    int num_of_threads_per_block = no_of_nodes;//正常限制为1024，这里先设置为节点数，然后再计算
    int max_threads_per_block = deviceProp.maxThreadsPerBlock;//硬件限制的
    //Make execution Parameters according to the number of nodes
    //Distribute threads across multiple Blocks if necessary
    if (no_of_nodes > max_threads_per_block)
    {
        num_of_blocks = (int)ceil(no_of_nodes/(double)max_threads_per_block);
        num_of_threads_per_block = max_threads_per_block; 
    }
    //setup execution parameters
    dim3  grid( num_of_blocks, 1, 1);//1维grid，有num_of_blocks个block
    dim3  block( num_of_threads_per_block, 1, 1);//1维threads


    //根据输入参数创建相应的全局内存数组，并做出相应操作
    //nodes，edges，3个状态相关数组及cost数组

    //首先是offsets数组，得注意这个大小为no_of_nodes+1
    // copy offsets to unified memory
    //checkCudaErrors定义与cudacommon
    uint32_t* graph_offsets = nullptr;
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
        //统一内存分配，之后数据第一次访问时将会按需自动迁移到访问设备
        checkCudaErrors(hipMallocManaged(&graph_offsets, sizeof(uint32_t)*(no_of_nodes+1)));
    }
    memcpy(graph_offsets, offsets, sizeof(uint32_t)*(no_of_nodes+1));

    if (uvm) {
        // do nothing, graph_offsets remains on CPU
    } else if (uvm_prefetch) { 
        //数据预取技术
        checkCudaErrors(hipMemPrefetchAsync(graph_offsets, sizeof(uint32_t)*(no_of_nodes+1), device));
    } else if (uvm_advise) {
        checkCudaErrors(hipMemAdvise(graph_offsets, sizeof(uint32_t)*(no_of_nodes+1), hipMemAdviseSetReadMostly, device));
        checkCudaErrors(hipMemAdvise(graph_offsets, sizeof(uint32_t)*(no_of_nodes+1), hipMemAdviseSetPreferredLocation, device));
    } else if (uvm_prefetch_advise) {
        checkCudaErrors(hipMemAdvise(graph_offsets, sizeof(uint32_t)*(no_of_nodes+1), hipMemAdviseSetReadMostly, device));
        checkCudaErrors(hipMemAdvise(graph_offsets, sizeof(uint32_t)*(no_of_nodes+1), hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemPrefetchAsync(graph_offsets, sizeof(uint32_t)*(no_of_nodes+1), device));
    } else {
        std::cerr << "unrecognized uvm flag, exiting..." << std::endl;
        exit(-1);
    }

    //然后是edges数组
    // copy edges to unified memory
    uint32_t* graph_edges = nullptr;
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
        checkCudaErrors(hipMallocManaged(&graph_edges, sizeof(uint32_t)*edge_list_size));
    }
    memcpy(graph_edges, edges, sizeof(uint32_t)*edge_list_size);
    if (uvm) {
        // Do nothing, graph_edges remains on CPU
    } else if (uvm_prefetch) {
        checkCudaErrors(hipMemPrefetchAsync(graph_edges, sizeof(uint32_t)*edge_list_size, device));
    } else if (uvm_advise) {
        checkCudaErrors(hipMemAdvise(graph_edges, sizeof(uint32_t)*edge_list_size, hipMemAdviseSetReadMostly, device));
        checkCudaErrors(hipMemAdvise(graph_edges, sizeof(uint32_t)*edge_list_size, hipMemAdviseSetPreferredLocation, device));
    } else if (uvm_prefetch_advise) {
        checkCudaErrors(hipMemAdvise(graph_edges, sizeof(uint32_t)*edge_list_size, hipMemAdviseSetReadMostly, device));
        checkCudaErrors(hipMemAdvise(graph_edges, sizeof(uint32_t)*edge_list_size, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemPrefetchAsync(graph_edges, sizeof(uint32_t)*edge_list_size, device));
    } else {
        std::cerr << "unrecognized uvm flag, exiting..." << std::endl;
        exit(-1);
    }

    //分配状态相关数组及cost并初始化
	// allocate and initalize the memory
    bool* graph_mask;//设备上使用
    bool* updating_graph_mask;//设备上使用
    bool* graph_visited;//设备上使用
    // allocate and initialize memory for result
    int *cost = nullptr;//设备上使用，执行完后才需要拷贝回主机
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
        //分配
        checkCudaErrors(hipMallocManaged(&graph_mask, sizeof(bool)*no_of_nodes));
        checkCudaErrors(hipMallocManaged(&updating_graph_mask, sizeof(bool)*no_of_nodes));
        checkCudaErrors(hipMallocManaged(&graph_visited, sizeof(bool)*no_of_nodes));
        hipError_t err = hipMallocManaged(&cost, sizeof(int)*no_of_nodes);
        if (err != hipSuccess) {
            checkCudaErrors(hipFree(graph_offsets));
            checkCudaErrors(hipFree(graph_edges));
            checkCudaErrors(hipFree(graph_mask));
            checkCudaErrors(hipFree(updating_graph_mask));
            checkCudaErrors(hipFree(graph_visited));
            checkCudaErrors(hipFree(cost));
            return FLT_MAX;
        }
    }

    // 异步初始化
    hipStream_t s1,s2,s3;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);
    hipStreamCreate(&s3);
    hipMemsetAsync(graph_mask, 0, sizeof(bool)*no_of_nodes, s1);
    hipMemsetAsync(updating_graph_mask, 0, sizeof(bool)*no_of_nodes, s2);
    hipMemsetAsync(graph_visited, 0, sizeof(bool)*no_of_nodes, s3);
    hipMemsetAsync(cost, -1, sizeof(int)*no_of_nodes);//使用默认流相当于s0
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipStreamDestroy(s3);

    //根据输入参数对4个状态相关数组做相应操作
    if (uvm) {
        // Do nothing. graph_mask, updating_graph_mask, and graph_visited unallocated
    } else if (uvm_advise) {
        checkCudaErrors(hipMemAdvise(graph_mask, sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(updating_graph_mask, sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(graph_visited, sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(cost, sizeof(int)*no_of_nodes, hipMemAdviseSetPreferredLocation, device));
    } else if (uvm_prefetch) {
        //相当于s0
        checkCudaErrors(hipMemPrefetchAsync(graph_mask, sizeof(bool)*no_of_nodes, device));
        hipStream_t s1, s2, s3;
        checkCudaErrors(hipStreamCreate(&s1));
        checkCudaErrors(hipStreamCreate(&s2));
        checkCudaErrors(hipStreamCreate(&s3));
        checkCudaErrors(hipMemPrefetchAsync(updating_graph_mask, sizeof(bool)*no_of_nodes, device, s1));
        checkCudaErrors(hipMemPrefetchAsync(graph_visited, sizeof(bool)*no_of_nodes, device, s2));
        checkCudaErrors(hipMemPrefetchAsync(cost, sizeof(int)*no_of_nodes, device, s3));
        checkCudaErrors(hipStreamDestroy(s1));
        checkCudaErrors(hipStreamDestroy(s2));
        checkCudaErrors(hipStreamDestroy(s3));
    } else if (uvm_prefetch_advise) {
        checkCudaErrors(hipMemAdvise(graph_mask, sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(updating_graph_mask, sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(graph_visited, sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(cost, sizeof(int)*no_of_nodes, hipMemAdviseSetPreferredLocation, device));
        hipStream_t s1, s2, s3;
        checkCudaErrors(hipStreamCreate(&s1));
        checkCudaErrors(hipStreamCreate(&s2));
        checkCudaErrors(hipStreamCreate(&s3));
        checkCudaErrors(hipMemPrefetchAsync(updating_graph_mask, sizeof(bool)*no_of_nodes, device, s1));
        checkCudaErrors(hipMemPrefetchAsync(graph_visited, sizeof(bool)*no_of_nodes, device, s2));
        checkCudaErrors(hipMemPrefetchAsync(cost, sizeof(int)*no_of_nodes, device, s3));
        checkCudaErrors(hipStreamDestroy(s1));
        checkCudaErrors(hipStreamDestroy(s2));
        checkCudaErrors(hipStreamDestroy(s3));
    }

	// bool if execution is over
    bool *over = nullptr;
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
        checkCudaErrors(hipMallocManaged(&over, sizeof(bool)));
    }

    //遍历所有节点，未访问就进入遍历
    int cnt = 0;//用于记录连通块数量
    int k = 0; //用于记录kernel执行次数
    while(true){
        //设备端查找未访问节点
        int32_t start_vertex = find_unvisited_vertex(graph_visited, no_of_nodes);
        if(start_vertex == -1) break;//全访问过
        //然后调用cuda_bfs执行
        // printf("visite node : %d\n", i);
        cuda_bfs_uvm(no_of_nodes, start_vertex, graph_offsets, graph_edges, 
            graph_mask, updating_graph_mask, graph_visited, cost, over,
            grid, block, kernel_time, k);
        ++cnt;
    }

    //统一拷贝回值
    // copy result from device to host
    // checkCudaErrors(hipEventRecord(tstart, 0));   
    if (uvm) {
        // Do nothing, cost stays on CPU
    } else if (uvm_advise) {
        checkCudaErrors(hipMemAdvise(cost, sizeof(int)*no_of_nodes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
    } else if (uvm_prefetch) {
        checkCudaErrors(hipMemPrefetchAsync(cost, sizeof(int)*no_of_nodes, hipCpuDeviceId));
    } else if (uvm_prefetch_advise) {
        checkCudaErrors(hipMemAdvise(cost, sizeof(int)*no_of_nodes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
        checkCudaErrors(hipMemPrefetchAsync(cost, sizeof(int)*no_of_nodes, hipCpuDeviceId));
    } else {
        std::cerr << "Unrecognized uvm option, exiting..." << std::endl;
        exit(-1);
    }
    
    auto end_t = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(end_t - start_t);
    double duration_t = double(duration.count()) * microseconds::period::num / microseconds::period::den;
    total_time += duration_t;
    
    // cleanup memory
	checkCudaErrors(hipFree(graph_offsets));
	checkCudaErrors(hipFree(graph_edges));
	checkCudaErrors(hipFree(graph_mask));
	checkCudaErrors(hipFree(updating_graph_mask));
	checkCudaErrors(hipFree(graph_visited));
	checkCudaErrors(hipFree(cost));
    checkCudaErrors(hipFree(over));

    //时间统输出一记录
    printf("total_time is %f seconds\n", total_time);
    printf("kernel_time is %f seconds\n", kernel_time);
    printf("graph_block is %d\n", cnt);
    printf("kernel_exe_times are %d\n", k);

    return total_time;
}


//设置设备端数组状态的内核函数
__global__ void setValueKernel(bool* mask, bool* visited, int* cost, int idx) {
    if (threadIdx.x == 0) { // 仅用 1 个线程
        mask[idx] = true;
        visited[idx] = true;
        cost[idx] = 0;
    }
}

//BFSGraph内再封装一层，进行从source出发具体的遍历
//BFSGraph内就进行创建相关状态数组和存结果数组，然后将这些数组传给cuda_bfs即可
//设备参数也应该传进来，毕竟是在外面进行内存创建,但拷贝还是发生在内层循环内，执行前要拷贝进去，执行完要拷贝出来
//返回值是这次遍历的总时间=传输时间+计算时间
void cuda_bfs(int no_of_nodes, int source,
    uint32_t *&d_offsets, uint32_t *&d_edges, bool* &d_graph_mask, bool* &d_updating_graph_mask, 
    bool* &d_graph_visited, int* &d_cost, bool *&d_over, 
    dim3 &grid, dim3 &block, double &kernel_time, double &transfer_time, int &k){
        //函数功能:
        //设置source相关状态，并将source涉及打状态相关数组值单独拷贝到设备端
        //执行do-while执行两个kernel
        
        hipEvent_t tstart, tstop;
        hipEventCreate(&tstart);
        hipEventCreate(&tstop);
        float elapsedTime = 0;
        hipEventRecord(tstart, 0);
        //写一个设备端值设置函数，无需拷贝
        // 启动 1 个线程块，内含 1 个线程
        setValueKernel<<<1, 1>>>(d_graph_mask, d_graph_visited, d_cost, source);
        hipEventRecord(tstop, 0);
        hipEventSynchronize(tstop);
        hipEventElapsedTime(&elapsedTime, tstart, tstop);
        kernel_time += elapsedTime * 1.e-3; // convert to seconds
    
        bool stop;
        //Call the Kernel untill all the elements of Frontier are not false
        do
        {
            //if no thread changes this value then the loop stops
            stop=false;
            hipMemcpy(d_over, &stop, sizeof(bool), hipMemcpyHostToDevice);
            
            hipEventRecord(tstart, 0);
            Kernel<<< grid, block, 0 >>>( d_offsets, d_edges, d_graph_mask, d_updating_graph_mask, d_graph_visited, d_cost, no_of_nodes);
            CHECK_CUDA_ERROR();
            hipEventRecord(tstop, 0);
            hipEventSynchronize(tstop);
            hipEventElapsedTime(&elapsedTime, tstart, tstop);
            kernel_time += elapsedTime * 1.e-3;
            
            // check if kernel execution generated an error
            hipEventRecord(tstart, 0);
            Kernel2<<< grid, block, 0 >>>( d_graph_mask, d_updating_graph_mask, d_graph_visited, d_over, no_of_nodes);
            CHECK_CUDA_ERROR();
            hipEventRecord(tstop, 0);
            hipEventSynchronize(tstop);
            hipEventElapsedTime(&elapsedTime, tstart, tstop);
            kernel_time += elapsedTime * 1.e-3;
            hipMemcpy( &stop, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
            k++;
        }
        while (stop);
    }

//BFSGraphUVM内再封装一层，进行从source出发具体的遍历
//BFSGraphUVM内就进行创建相关状态数组和存结果数组，然后将这些数组传给cuda_bfs_uvm即可
//设备参数也应该传进来，毕竟是在外面进行内存创建,但拷贝还是发生在内层循环内，执行前要拷贝进去，执行完要拷贝出来
void cuda_bfs_uvm(int no_of_nodes, int source, uint32_t *&graph_offsets, uint32_t *&graph_edges,
     bool* &graph_mask, bool* &updating_graph_mask, bool* &graph_visited, int* &cost, bool *&over, 
     dim3 &grid, dim3 &block, double &kernel_time, int &k){
        //内层循环功能:
        //设置source相关状态，无需拷贝，这里将会自动管理内存
        //执行do-while执行两个kernel
        //因为使用uvm，所以少了数据管理

        //set the source node as true in the mask and cost
	    graph_mask[source]=true;
	    graph_visited[source]=true;
        cost[source]=0;

        // events for timing
        hipEvent_t tstart, tstop;
        checkCudaErrors(hipEventCreate(&tstart));
        checkCudaErrors(hipEventCreate(&tstop));
        float elapsedTime = 0; 
        // // 启动 1 个线程块，内含 1 个线程
        // setValueKernel<<<1, 1>>>(graph_mask, graph_visited, cost, source);
        // CHECK_CUDA_ERROR();
        // hipEventRecord(tstop, 0);
        // hipEventSynchronize(tstop);
        // hipEventElapsedTime(&elapsedTime, tstart, tstop);
        // kernel_time += elapsedTime * 1.e-3;
        
        bool stop;
        //Call the Kernel until all the elements of Frontier are not false
        do
        {
            stop = false;
            *over = stop;

            checkCudaErrors(hipEventRecord(tstart, 0));
            Kernel<<< grid, block, 0 >>>(graph_offsets, graph_edges, graph_mask, updating_graph_mask, graph_visited, cost, no_of_nodes);
            checkCudaErrors(hipEventRecord(tstop, 0));
            checkCudaErrors(hipEventSynchronize(tstop));
            checkCudaErrors(hipEventElapsedTime(&elapsedTime, tstart, tstop));
            kernel_time += elapsedTime * 1.e-3;
            CHECK_CUDA_ERROR();

            // check if kernel execution generated an error
            checkCudaErrors(hipEventRecord(tstart, 0));
            Kernel2<<< grid, block, 0 >>>(graph_mask, updating_graph_mask, graph_visited, over, no_of_nodes);
            checkCudaErrors(hipEventRecord(tstop, 0));
            checkCudaErrors(hipEventSynchronize(tstop));
            checkCudaErrors(hipEventElapsedTime(&elapsedTime, tstart, tstop));
            kernel_time += elapsedTime * 1.e-3;
            CHECK_CUDA_ERROR();

            stop = *over;
            k++;
        }
        while (stop);
    }

void checkCudaFeatureAvailability(OptionParser &op) {
    int device = 0;
    checkCudaErrors(hipGetDevice(&device));
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, device));
    
    // Check UVM availability
    if (op.getOptionBool("uvm") || op.getOptionBool("uvm-advise") ||
            op.getOptionBool("uvm-prefetch") || op.getOptionBool("uvm-prefetch-advise")) {
        if (!deviceProp.unifiedAddressing) {
            std::cerr << "device doesn't support unified addressing, exiting..." << std::endl;
            safe_exit(-1);
        }
    }

    // Check Cooperative Group availability
    if (op.getOptionBool("coop")) {
        if (!deviceProp.cooperativeLaunch) {
            std::cerr << "device doesn't support cooperative kernels, exiting..." << std::endl;
            safe_exit(-1);
        }
    }

    // Check Dynamic Parallelism availability
    if (op.getOptionBool("dyn")) {
        int runtimeVersion = 0;
        checkCudaErrors(hipRuntimeGetVersion(&runtimeVersion));
        if (runtimeVersion < 5000) {
            std::cerr << "CUDA runtime version less than 5.0, doesn't support \
                dynamic parallelism, exiting..." << std::endl;
            safe_exit(-1);
        }
    }

    // Check CUDA Graphs availability
    if (op.getOptionBool("graph")) {
        int runtimeVersion = 0;
        checkCudaErrors(hipRuntimeGetVersion(&runtimeVersion));
        if (runtimeVersion < 10000) {
            std::cerr << "CUDA runtime version less than 10.0, doesn't support \
                CUDA Graph, exiting..." << std::endl;
            safe_exit(-1);
        }
    }
}